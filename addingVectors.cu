/*
    steps: 183.82.46.36
    kmit
    udaan@123$
    cd ngit_demo
    cd KMIT_20BD5a0502
    Open editor and write the code...
*/

// Addition of vectors..
// CASE-1: n blocks and one thread..

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void addVectors(int *a, int *b, int *c){
    for(int tid=0; tid<N; tid++)
        c[tid] = a[tid] + b[tid];
}

int main(){
    // Declare the arrays.. and fill with the default values..
    int a[N] = {}, b[N] = {}, c[N] = {};
    int *dev_a, *dev_b, *dev_c;

    // allocate memory on device..
    hipMalloc((void**) &dev_a, N*sizeof(int));
    hipMalloc((void**) &dev_b, N*sizeof(int));
    hipMalloc((void**) &dev_c, N*sizeof(int));

    // Fill with some values..
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = N-i;
    }

    // copy the arrays to the device..
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    // launch the kernel
    addVectors<<<1, 1>>>(dev_a, dev_b, dev_c);

    // copy the results back..
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    // printing the results..
    for(int i=0; i<N; i++)
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
}
